
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N (1024 * 1024)

__global__ void kernel (float * data)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // current thread number
    float x = 2.0f * 3.1415926f * (float) idx / (float) N; // argument value
    data[idx] = sinf(sqrtf(x)); // find a value and write it down to an array
}

// - allocate a private thread to each array element (total N)
// - this private threads calculate required values
// - each thread has unique id

int main (int argc, char * argv[])
{
    float * a = new float [N]; // allocate host memory (dynamic array)
    float * dev = NULL; // allocate device memory

    // allocate GPU memory for N elements
    hipMalloc((void**) & dev, N * sizeof(float));

    // launch N blocks by 512 threads
    // @variable: kernel - performed function per thread
    // @variable: dev - data array
    kernel <<< dim3((N / 512), 1), dim3(512, 1) >>> (dev);

    // copy the results from GPU (DRAM) to CPU (N elements)
    hipMemcpy(a, dev, N * sizeof(float), hipMemcpyDeviceToHost);

    // free up memory
    hipFree(dev);

    // print results
    for (int idx = 0; idx < N; idx++) 
    {
        printf("a[%d] = %.5f\n", idx, a[idx]);
    }

    return 0;
}